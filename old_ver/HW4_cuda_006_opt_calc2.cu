#include <iostream>
#include <algorithm>
#include <sstream>
#include <fstream>
#include <vector>

#include <stdio.h>
#include <stdlib.h>
#include <stdarg.h>

#include <hip/hip_runtime.h>
#include <unistd.h>

//#define _DEBUG_
//#define _TIME_MEASURE_

#ifdef _DEBUG_
    #include <string>
    #include <sstream>

    int __print_step = 0;
    
    void __pt_log(const char *h_, const char *f_, ...){
        std::stringstream ss;
        ss << h_ << f_ << '\n';
        std::string format = ss.str();

        va_list va;
        va_start(va, f_);
            vprintf(format.c_str(), va);
        va_end(va);
        __print_step++;
    }

    #define VA_ARGS(...) , ##__VA_ARGS__
    #define LOG(f_, ...) __pt_log(\
                                    "[LOG] Step %3d: ", (f_), \
                                     __print_step VA_ARGS(__VA_ARGS__))
#else
    #define LOG(f_, ...)
#endif


#ifdef _TIME_MEASURE_

    #define PRECISION 1000

    #include <chrono>
    #include <map>

    using hr_clock = std::chrono::high_resolution_clock;

    struct __timer{
        bool state;
        double total;
        std::chrono::time_point<hr_clock> start;
        __timer() : state(false), total(0){}
    };

    std::map<std::string, struct __timer> __t_map;
    inline void __ms_tic(std::string tag, bool cover=true){
        try{
            __timer &t = __t_map[tag];
            if(!cover && t.state) 
                throw std::string("the timer has already started");
            t.state = true;
            t.start = std::chrono::high_resolution_clock::now();
        }catch(std::string msg){
            msg += std::string(": %s");
            LOG(msg.c_str(), tag.c_str());
        }
    }

    inline void __ms_toc(std::string tag, bool restart=false){
        auto end = std::chrono::high_resolution_clock::now();
        try{
            __timer &t = __t_map[tag];
            if(!t.state)
                throw std::string("the timer is inactive");
            t.state = restart;
            std::chrono::duration<double> d = end-t.start;
            t.total += d.count() * PRECISION;
            t.start = end;
        }catch(std::string msg){
            msg += std::string(": %s");
            LOG(msg.c_str(), tag.c_str());
        }
    }

    inline void __log_all(){
        LOG("%-30s %-30s", "Timers", "Elapsed time");
        for(auto it=__t_map.begin(); it!=__t_map.end(); ++it)
            LOG("%-30s %.6lf ms", it->first.c_str(), it->second.total);
    }

    #define TIC(tag, ...) __ms_tic((tag))
    #define TOC(tag, ...) __ms_toc((tag))
    #define GET(tag) __t_map[tag].total;
    #define _LOG_ALL() __log_all()
#else
    #define TIC(tag, ...)
    #define TOC(tag, ...)
    #define GET(tag) 0
    #define _LOG_ALL()
#endif



#define CEIL(a, b) ((a) + (b) -1)/(b)
#define INF 1000000000

int **Dist;
int *data;
int block_size;
int vert, edge;
int vert2;

inline void init(){
    vert2 = vert*vert;
    Dist = new int*[vert];
    hipHostMalloc(&data, vert2*sizeof(int), hipHostMallocDefault);

    std::fill(data, data + vert2, INF);

    for(int i=0;i<vert;++i){
        Dist[i] = data + i*vert;
        Dist[i][i] = 0;
    }
}

inline void finalize(){
    delete[] Dist;
    hipFree(data);
}

void parse_string(std::stringstream &ss, std::vector<int> &int_list){

    std::string str = ss.str();
    const char *buf = str.c_str();
    size_t sz = str.size();

    int item = 0;
    for (size_t i = 0; i < sz; ++i){
        switch (buf[i]){
            case '\n':
                int_list.push_back(item);
                item = 0;
                break;
            case ' ':
                int_list.push_back(item);
                item = 0;
                break;
            default:
                item = 10*item + buf[i] - '0';
                break;
        }    
    }
}

void dump_from_file_and_init(const char *file){
    TIC("init/read_file");
    std::ifstream fin(file);
    std::stringstream ss;

    ss << fin.rdbuf();
    ss >> vert >> edge;

    TOC("init/read_file");

    TIC("init/parse_int");

    std::vector<int> int_list;
    int_list.reserve(edge * 3+2);

    init();

    parse_string(ss, int_list);

    TOC("init/parse_int");
    TIC("init/init_mat");

    for(auto e = int_list.begin()+2; e != int_list.end(); e+=3){
        Dist[*e][*(e+1)] = *(e+2);
    }

    fin.close();

    TOC("init/init_mat");
}

void dump_to_file(const char *file){
    FILE *fout = fopen(file, "w");
    fwrite(data, sizeof(int) * vert2, 1, fout);
    fclose(fout);
}

template<int block_size>
__global__ void phase_one(int32_t* const dist, const int round, const int width, const int vert, const int br){

    __shared__ int s[block_size][block_size];

    const int c = br + threadIdx.y;
    const int r = br + threadIdx.x;
    const int cell = c * width + r;

    const bool mb = (c < vert && r < vert);
    s[threadIdx.y][threadIdx.x] = (mb) ? dist[cell] : INF;

    if( !mb ) return;
    
    __syncthreads();
    int mn = s[threadIdx.y][threadIdx.x];
    int n;
    for(int k=0;k<block_size;++k){
        n = s[threadIdx.y][k] + s[k][threadIdx.x];
        if(n < mn){
            s[threadIdx.y][threadIdx.x] = n;
            mn = n;
        }
        __syncthreads();
    }

    dist[cell] = mn;
}

template<int block_size>
__global__ void phase_two(int32_t* const dist, const int round, const int width, const int vert, const int br){

    if(blockIdx.x == round) return;

    __shared__ int s_m[block_size][block_size];
    __shared__ int s_c[block_size][block_size];

    int mc, mr;
    int cc, cr;

    if(blockIdx.y == 0){
        mc = br + threadIdx.y;
        mr = block_size * blockIdx.x + threadIdx.x;
        cc = mc;
        cr = br + threadIdx.x;
    }else{
        mc = block_size * blockIdx.x + threadIdx.y;
        mr = br + threadIdx.x;
        cc = br + threadIdx.y;
        cr = mr;
    }

    const int m_cell = mc * width + mr;
    //const int c_cell = cc * width + cr;

    const bool mb = (mc < vert && mr < vert);
    const bool cb = (cc < vert && cr < vert);

    s_m[threadIdx.y][threadIdx.x] = (mb) ? dist[m_cell] : INF;
    s_c[threadIdx.y][threadIdx.x] = (cb) ? dist[cc * width + cr] : INF;

    if( !mb ) return;

    __syncthreads();

    int mn = s_m[threadIdx.y][threadIdx.x];
    int n;

    if(blockIdx.y == 0){
        for(int k=0;k<block_size;++k){
            n = s_c[threadIdx.y][k] + s_m[k][threadIdx.x];
            if(n < mn){
                s_m[threadIdx.y][threadIdx.x] = n;
                mn = n;
            }
            __syncthreads();
        }
    }else{
        for(int k=0;k<block_size;++k){
            n = s_m[threadIdx.y][k] + s_c[k][threadIdx.x];
            if(n < mn){
                s_m[threadIdx.y][threadIdx.x] = n;
                mn = n;
            }
            __syncthreads();
        }
    }

    dist[m_cell] = mn;
}

template<int block_size>
__global__ void phase_three(int32_t* const dist, const int round, const int width, const int vert, const int br){
    
    if(blockIdx.x == round || blockIdx.y == round) return;

    __shared__ int s_l[block_size][block_size];
    __shared__ int s_r[block_size][block_size];

    const int mc = block_size * blockIdx.y + threadIdx.y;
    const int mr = block_size * blockIdx.x + threadIdx.x;
    const int lr = br + threadIdx.x;
    const int rc = br + threadIdx.y;

    s_l[threadIdx.y][threadIdx.x] = (mc < vert && lr < vert) ? dist[mc * width + lr] : INF;
    s_r[threadIdx.y][threadIdx.x] = (rc < vert && mr < vert) ? dist[rc * width + mr] : INF;

    if( !(mc < vert && mr < vert) ) return;

    const int m_cell = mc * width + mr;
    int mn = dist[m_cell];

    __syncthreads();

    int n;
    for(int k=0;k<block_size;++k){
        n = s_l[threadIdx.y][k] + s_r[k][threadIdx.x];
        if( n < mn) mn = n;
    }

    dist[m_cell] = mn;
}

template<int BLOCK_SIZE> void block_FW(){
    int Round = CEIL(vert, BLOCK_SIZE);
    size_t vert_bytes = vert * sizeof(int);

    int32_t *device_ptr;
    size_t pitch_bytes;

    dim3 p2b(Round, 2, 1);
    dim3 p3b(Round, Round, 1);

    dim3 dimt(BLOCK_SIZE, BLOCK_SIZE, 1);


    hipMallocPitch(&device_ptr, &pitch_bytes, vert_bytes, vert);

    hipMemcpy2DAsync(device_ptr, pitch_bytes, data, vert_bytes,
                    vert_bytes, vert, hipMemcpyHostToDevice);

    int pitch = pitch_bytes / sizeof(int);
    hipDeviceSynchronize();

    int br = 0;
    for(int r=0;r<Round;++r){
        phase_one< BLOCK_SIZE ><<< 1 , dimt >>>(device_ptr, r, pitch, vert, br);
        phase_two< BLOCK_SIZE ><<< p2b , dimt >>>(device_ptr, r, pitch, vert, br);
        phase_three< BLOCK_SIZE ><<< p3b , dimt >>>(device_ptr, r, pitch, vert, br);
        br += BLOCK_SIZE;
    }

    hipMemcpy2D(data, vert_bytes, device_ptr, pitch_bytes, vert_bytes, vert, hipMemcpyDeviceToHost);

    hipFree(device_ptr);
}

int main(int argc, char **argv){

    TIC("init");
    dump_from_file_and_init(argv[1]);

    TOC("init");


    TIC("block");

    block_size = std::atoi(argv[3]);
    switch(block_size){
        case 8:
            block_FW<8>();
            break;
        case 16:
            block_FW<16>();
            break;
        case 24:
            block_FW<24>();
            break;
        case 32:
            block_FW<32>();
            break;
    }

    TOC("block");



    TIC("write_file");

    dump_to_file(argv[2]);

    TOC("write_file");


    TIC("finalize");

    finalize();

    TOC("finalize");

    _LOG_ALL();
    return 0;
}






