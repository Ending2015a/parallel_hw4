#include <iostream>
#include <algorithm>
#include <sstream>
#include <fstream>
#include <vector>

#include <stdio.h>
#include <stdlib.h>
#include <stdarg.h>

#include <hip/hip_runtime.h>
#include <unistd.h>

//#define _DEBUG_
//#define _TIME_MEASURE_

#ifdef _DEBUG_
    #include <string>
    #include <sstream>

    int __print_step = 0;
    
    void __pt_log(const char *h_, const char *f_, ...){
        std::stringstream ss;
        ss << h_ << f_ << '\n';
        std::string format = ss.str();

        va_list va;
        va_start(va, f_);
            vprintf(format.c_str(), va);
        va_end(va);
        __print_step++;
    }

    #define VA_ARGS(...) , ##__VA_ARGS__
    #define LOG(f_, ...) __pt_log(\
                                    "[LOG] Step %3d: ", (f_), \
                                     __print_step VA_ARGS(__VA_ARGS__))
#else
    #define LOG(f_, ...)
#endif


#ifdef _TIME_MEASURE_

    #define PRECISION 1000

    #include <chrono>
    #include <map>

    using hr_clock = std::chrono::high_resolution_clock;

    struct __timer{
        bool state;
        double total;
        std::chrono::time_point<hr_clock> start;
        __timer() : state(false), total(0){}
    };

    std::map<std::string, struct __timer> __t_map;
    inline void __ms_tic(std::string tag, bool cover=true){
        try{
            __timer &t = __t_map[tag];
            if(!cover && t.state) 
                throw std::string("the timer has already started");
            t.state = true;
            t.start = std::chrono::high_resolution_clock::now();
        }catch(std::string msg){
            msg += std::string(": %s");
            LOG(msg.c_str(), tag.c_str());
        }
    }

    inline void __ms_toc(std::string tag, bool restart=false){
        auto end = std::chrono::high_resolution_clock::now();
        try{
            __timer &t = __t_map[tag];
            if(!t.state)
                throw std::string("the timer is inactive");
            t.state = restart;
            std::chrono::duration<double> d = end-t.start;
            t.total += d.count() * PRECISION;
            t.start = end;
        }catch(std::string msg){
            msg += std::string(": %s");
            LOG(msg.c_str(), tag.c_str());
        }
    }

    inline void __log_all(){
        LOG("%-30s %-30s", "Timers", "Elapsed time");
        for(auto it=__t_map.begin(); it!=__t_map.end(); ++it)
            LOG("%-30s %.6lf ms", it->first.c_str(), it->second.total);
    }

    #define TIC(tag, ...) __ms_tic((tag))
    #define TOC(tag, ...) __ms_toc((tag))
    #define GET(tag) __t_map[tag].total;
    #define _LOG_ALL() __log_all()
#else
    #define TIC(tag, ...)
    #define TOC(tag, ...)
    #define GET(tag) 0
    #define _LOG_ALL()
#endif


#define MAX(a, b) ((a)>(b)?(a):(b))
#define MIN(a, b) ((a)<(b)?(a):(b))
#define CEIL(a, b) ((a) + (b) -1)/(b)
#define INF 1000000000
#define MAX_BLOCK_SIZE 32


int **Dist;
int *data;
int block_size;
int vert, edge;
int vert2;

inline void init(){
    vert2 = vert*vert;
    Dist = new int*[vert];
    hipHostMalloc(&data, vert2*sizeof(int), hipHostMallocDefault);

    std::fill(data, data + vert2, INF);

    for(int i=0;i<vert;++i){
        Dist[i] = data + i*vert;
        Dist[i][i] = 0;
    }
}

inline void finalize(){
    delete[] Dist;
    hipFree(data);
}

void parse_string(std::stringstream &ss, int *int_list){

    std::string str = ss.str();
    char *buf = (char*)str.c_str();
    size_t sz = str.size();
    char *end = buf+sz;

    int item = 0;
    for (; buf < end; ++buf){
        switch (*buf){
            case '\n':
            case ' ':
                *int_list=item;
                ++int_list;
                item = 0;
                break;
            default:
                item = 10*item + (*buf - '0');
                break;
        }
    }
}

void dump_from_file_and_init(const char *file){
    TIC("init/read_file");
    std::ifstream fin(file);
    std::stringstream ss;

    ss << fin.rdbuf();
    ss >> vert >> edge;

    TOC("init/read_file");

    TIC("init/parse_int");

    //std::vector<int> int_list;

    int sz = edge*3+2;
    
    int *int_list = new int[sz];
    //int_list.reserve(edge * 3+2);

    init();

    parse_string(ss, int_list);

    TOC("init/parse_int");
    TIC("init/init_mat");

    int *end = int_list + sz;
    for(int* e = int_list+2; e < end ; e+=3){
        Dist[*e][*(e+1)] = *(e+2);
    }

    fin.close();

    delete[] int_list;
    TOC("init/init_mat");
}

void dump_to_file(const char *file){
    FILE *fout = fopen(file, "w");
    fwrite(data, sizeof(int) * vert2, 1, fout);
    fclose(fout);
}

template<int block_size>
__global__ void phase_one(int32_t* const dist, const int round, const int width, const int vert, const int br){

    __shared__ int s[block_size][block_size];

    const int c = br + threadIdx.y;
    const int r = br + threadIdx.x;
    const int cell = c * width + r;

    const bool mb = (c < vert && r < vert);
    s[threadIdx.y][threadIdx.x] = (mb) ? dist[cell] : INF;

    if( !mb ) return;

    int o = s[threadIdx.y][threadIdx.x];
    int n;
    for(int k=0;k<block_size;++k){

        __syncthreads();

        n = s[threadIdx.y][k] + s[k][threadIdx.x];
        if(n < s[threadIdx.y][threadIdx.x]){
            s[threadIdx.y][threadIdx.x] = n;
        }
    }

    if(s[threadIdx.y][threadIdx.x] < o)
        dist[cell] = s[threadIdx.y][threadIdx.x];
}

__global__ void phase_one(int32_t* const dist, const int round, const int width, const int vert, const int br){

    __shared__ int s[MAX_BLOCK_SIZE][MAX_BLOCK_SIZE];

    const int c = br + threadIdx.y;
    const int r = br + threadIdx.x;
    const int cell = c * width + r;

    const bool mb = (c < vert && r < vert);
    s[threadIdx.y][threadIdx.x] = (mb) ? dist[cell] : INF;

    if( !mb ) return;
    
    int upper = MIN(vert - br, blockDim.x);

    int o = s[threadIdx.y][threadIdx.x];
    int n;
    for(int k=0;k<upper;++k){

        __syncthreads();

        n = s[threadIdx.y][k] + s[k][threadIdx.x];
        if(n < s[threadIdx.y][threadIdx.x]){
            s[threadIdx.y][threadIdx.x] = n;
        }
    }

    if( s[threadIdx.y][threadIdx.x] < o)
        dist[cell] = s[threadIdx.y][threadIdx.x];
}

template<int block_size>
__global__ void phase_two(int32_t* const dist, const int round, const int width, const int vert, const int br){

    if(blockIdx.x == round) return;

    __shared__ int s_m[block_size][block_size];
    __shared__ int s_c[block_size][block_size];

    int mc, mr;
    int cc, cr;

    if(blockIdx.y == 0){
        mc = br + threadIdx.y;
        mr = block_size * blockIdx.x + threadIdx.x;
        cc = mc;
        cr = br + threadIdx.x;
    }else{
        mc = block_size * blockIdx.x + threadIdx.y;
        mr = br + threadIdx.x;
        cc = br + threadIdx.y;
        cr = mr;
    }

    const int m_cell = mc * width + mr;

    const bool mb = (mc < vert && mr < vert);
    const bool cb = (cc < vert && cr < vert);

    s_m[threadIdx.y][threadIdx.x] = (mb) ? dist[m_cell] : INF;
    s_c[threadIdx.y][threadIdx.x] = (cb) ? dist[cc * width + cr] : INF;

    if( !mb ) return;

    int o = s_m[threadIdx.y][threadIdx.x];
    int n;
    if(blockIdx.y == 0){
        for(int k=0;k<block_size;++k){

            __syncthreads();

            n = s_c[threadIdx.y][k] + s_m[k][threadIdx.x];
            if(n < s_m[threadIdx.y][threadIdx.x]){
                s_m[threadIdx.y][threadIdx.x] = n;
            }
        }
    }else{
        for(int k=0;k<block_size;++k){

            __syncthreads();

            n = s_m[threadIdx.y][k] + s_c[k][threadIdx.x];
            if(n < s_m[threadIdx.y][threadIdx.x]){
                s_m[threadIdx.y][threadIdx.x] = n;
            }
        }
    }
    if(s_m[threadIdx.y][threadIdx.x] < o)
        dist[m_cell] = s_m[threadIdx.y][threadIdx.x];
}

__global__ void phase_two(int32_t* const dist, const int round, const int width, const int vert, const int br){

    if(blockIdx.x == round) return;

    __shared__ int s_m[MAX_BLOCK_SIZE][MAX_BLOCK_SIZE];
    __shared__ int s_c[MAX_BLOCK_SIZE][MAX_BLOCK_SIZE];

    int mc, mr;
    int cc, cr;

    if(blockIdx.y == 0){
        mc = br + threadIdx.y;
        mr = blockDim.x * blockIdx.x + threadIdx.x;
        cc = mc;
        cr = br + threadIdx.x;
    }else{
        mc = blockDim.x * blockIdx.x + threadIdx.y;
        mr = br + threadIdx.x;
        cc = br + threadIdx.y;
        cr = mr;
    }

    const int m_cell = mc * width + mr;

    const bool mb = (mc < vert && mr < vert);
    const bool cb = (cc < vert && cr < vert);

    s_m[threadIdx.y][threadIdx.x] = (mb) ? dist[m_cell] : INF;
    s_c[threadIdx.y][threadIdx.x] = (cb) ? dist[cc * width + cr] : INF;

    if( !mb ) return;

    int upper = MIN(vert-br, blockDim.x);

    int o = s_m[threadIdx.y][threadIdx.x];
    int n;
    if(blockIdx.y == 0){
        for(int k=0;k<upper;++k){

            __syncthreads();

            n = s_c[threadIdx.y][k] + s_m[k][threadIdx.x];
            if(n < s_m[threadIdx.y][threadIdx.x]){
                s_m[threadIdx.y][threadIdx.x] = n;
            }
        }
    }else{
        for(int k=0;k<upper;++k){

            __syncthreads();

            n = s_m[threadIdx.y][k] + s_c[k][threadIdx.x];
            if(n < s_m[threadIdx.y][threadIdx.x]){
                s_m[threadIdx.y][threadIdx.x] = n;
            }
        }
    }

    if(s_m[threadIdx.y][threadIdx.x] < o)
        dist[m_cell] = s_m[threadIdx.y][threadIdx.x];
}



template<int block_size>
__global__ void phase_three(int32_t* const dist, const int round, const int width, const int vert, const int br){
    
    if(blockIdx.x == round || blockIdx.y == round) return;

    __shared__ int s_l[block_size][block_size];
    __shared__ int s_r[block_size][block_size];

    const int mc = block_size * blockIdx.y + threadIdx.y;
    const int mr = block_size * blockIdx.x + threadIdx.x;
    const int lr = br + threadIdx.x;
    const int rc = br + threadIdx.y;

    s_l[threadIdx.y][threadIdx.x] = (mc < vert && lr < vert) ? dist[mc * width + lr] : INF;
    s_r[threadIdx.y][threadIdx.x] = (rc < vert && mr < vert) ? dist[rc * width + mr] : INF;

    if( mc >= vert || mr >= vert ) return;

    const int m_cell = mc * width + mr;
    __syncthreads();

    int o = dist[m_cell];

    int n;
    int mn=s_l[threadIdx.y][0] + s_r[0][threadIdx.x];
    for(int k=1;k<block_size;++k){
        n = s_l[threadIdx.y][k] + s_r[k][threadIdx.x];
        if(n < mn) mn = n;
    }

    if(mn < o)
        dist[m_cell] = mn;
}

__global__ void phase_three(int32_t* const dist, const int round, const int width, const int vert, const int br){
    
    if(blockIdx.x == round || blockIdx.y == round) return;

    __shared__ int s_l[MAX_BLOCK_SIZE][MAX_BLOCK_SIZE];
    __shared__ int s_r[MAX_BLOCK_SIZE][MAX_BLOCK_SIZE];

    const int mc = blockDim.x * blockIdx.y + threadIdx.y;
    const int mr = blockDim.x * blockIdx.x + threadIdx.x;
    const int lr = br + threadIdx.x;
    const int rc = br + threadIdx.y;

    s_l[threadIdx.y][threadIdx.x] = (mc < vert && lr < vert) ? dist[mc * width + lr] : INF;
    s_r[threadIdx.y][threadIdx.x] = (rc < vert && mr < vert) ? dist[rc * width + mr] : INF;

    if( mc >= vert || mr >= vert ) return;

    const int m_cell = mc * width + mr;
    int upper = MIN(vert - br, blockDim.x);
    __syncthreads();

    int o = dist[m_cell];
    int n;
    int mn = s_l[threadIdx.y][0] + s_r[0][threadIdx.x];
    for(int k=1;k<upper;++k){
        n = s_l[threadIdx.y][k] + s_r[k][threadIdx.x];
        if( n < mn ) mn = n;
    }

    if( mn < o )
        dist[m_cell] = mn;
}



template<int BLOCK_SIZE> 
void block_FW(){
    int Round = CEIL(vert, BLOCK_SIZE);
    size_t vert_bytes = vert * sizeof(int);

    int32_t *device_ptr;
    size_t pitch_bytes;

    dim3 p2b(Round, 2, 1);
    dim3 p3b(Round, Round, 1);

    dim3 dimt(BLOCK_SIZE, BLOCK_SIZE, 1);


    hipMallocPitch(&device_ptr, &pitch_bytes, vert_bytes, vert);

    hipMemcpy2D(device_ptr, pitch_bytes, data, vert_bytes,
                    vert_bytes, vert, hipMemcpyHostToDevice);

    int pitch = pitch_bytes / sizeof(int);
    //cudaDeviceSynchronize();

    int br = 0;
    for(int r=0;r<Round;++r){
        phase_one< BLOCK_SIZE ><<< 1 , dimt >>>(device_ptr, r, pitch, vert, br);
        phase_two< BLOCK_SIZE ><<< p2b , dimt >>>(device_ptr, r, pitch, vert, br);
        phase_three< BLOCK_SIZE ><<< p3b , dimt >>>(device_ptr, r, pitch, vert, br);
        br += BLOCK_SIZE;
    }

    hipDeviceSynchronize();
    hipMemcpy2D(data, vert_bytes, device_ptr, pitch_bytes, vert_bytes, vert, hipMemcpyDeviceToHost);

    hipFree(device_ptr);
}

void block_FW(){
    int Round = CEIL(vert, block_size);
    size_t vert_bytes = vert * sizeof(int);

    int32_t *device_ptr;
    size_t pitch_bytes;

    dim3 p2b(Round, 2, 1);
    dim3 p3b(Round, Round, 1);

    dim3 dimt(block_size, block_size, 1);



    hipMallocPitch(&device_ptr, &pitch_bytes, vert_bytes, vert);

    hipMemcpy2DAsync(device_ptr, pitch_bytes, data, vert_bytes,
                    vert_bytes, vert, hipMemcpyHostToDevice);

    hipDeviceSynchronize();

    int pitch = pitch_bytes / sizeof(int);
    int br = 0;
    for(int r=0;r<Round;++r){
        phase_one<<< 1 , dimt >>>(device_ptr, r, pitch, vert, br);
        phase_two<<< p2b , dimt >>>(device_ptr, r, pitch, vert, br);
        phase_three<<< p3b , dimt >>>(device_ptr, r, pitch, vert, br);
        br += block_size;
    }

    hipMemcpy2D(data, vert_bytes, device_ptr, pitch_bytes, vert_bytes, vert, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    hipFree(device_ptr);
}


int main(int argc, char **argv){

    TIC("init");
    dump_from_file_and_init(argv[1]);

    TOC("init");


    TIC("block");

    block_size = std::atoi(argv[3]);
    switch(block_size){
        case 8:
            block_FW<8>();
            break;
        case 16:
            block_FW<16>();
            break;
        case 24:
            block_FW<24>();
            break;
        case 32:
            block_FW<32>();
            break;
        default:
            block_FW();
            break;
    }

    TOC("block");



    TIC("write_file");

    dump_to_file(argv[2]);

    TOC("write_file");


    TIC("finalize");

    finalize();

    TOC("finalize");

    _LOG_ALL();
    return 0;
}






